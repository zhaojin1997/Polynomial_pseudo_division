#include "hip/hip_runtime.h"
#include "add.h"
vector<float> addInteger(vector<float> a, vector<float> b) {
	int lena = a.size();
	int lenb = b.size();
	int aflag = a[0];
	int bflag = b[0];
	if (aflag < 0) {
		a[0] = -a[0];
	}if (bflag < 0) {
		b[0] = -b[0];
	}
	int lendev;
	vector<float> tmp;
	if (lena > lenb) {
		lendev = lena - lenb;
		for (int i = 0; i < lendev; i++) {
			tmp.insert(tmp.end(), 0);
		}
		tmp.insert(tmp.end(), b.begin(), b.end());
		b = tmp;
	}
	else {
		lendev = lenb - lena;
		for (int i = 0; i < lendev; i++) {
			tmp.insert(tmp.end(), 0);
		}
		tmp.insert(tmp.end(), a.begin(), a.end());
		a = tmp;
	}
	vector<float> tmpresult(a.size() + 1);
	bool changeflag = false;
	if (aflag > 0 && bflag > 0) {
		tmpresult = addmain(a, b);
	}
	else if (aflag< 0 && bflag>0) {
		if (lena > lenb) {
			tmpresult = submain(a, b);
			//tmpresult[0] = -tmpresult[0];
			changeflag = true;
		}
		else if (lena<lenb) {
			tmpresult = submain(b, a);
		}
		else if (lena == lenb) {
			bool flag = false;
			for (int i = 0; i < lena; i++) {
				if (a[i] > b[i]) {
					flag = true;
					break;
				}
			}
			if (flag == true) {
				tmpresult = submain(a, b);
				changeflag = true;
				//tmpresult[0] = -tmpresult[0];
			}
			else {
				tmpresult = submain(b, a);
			}
		}

	}
	else if (aflag > 0 && bflag < 0) {
		if (lena > lenb) {
			tmpresult = submain(a, b);
		}
		else if (lena < lenb) {
			tmpresult = submain(b, a);
			changeflag = true;
			//tmpresult[0] = -tmpresult[0];
		}
		else if (lena == lenb) {
			bool flag = false;
			for (int i = 0; i < lena; i++) {
				if (a[i] > b[i]) {
					flag = true;
					break;
				}
			}
			if (flag == true) {
				tmpresult = submain(a, b);
			}
			else {
				tmpresult = submain(b, a);
				changeflag = true;
				//tmpresult[0] = -tmpresult[0];
			}
		}
	}
	else if (aflag < 0 && bflag < 0) {
		tmpresult = addmain(a, b);
		changeflag = true;
		//tmpresult[0] = -tmpresult[0];
	}
	vector<int> c;
	for (int i = 0; i <tmpresult.size(); i++) {
		int a = int(tmpresult[i]);
		c.insert(c.end(), a);
	}
	const int base = 10;
	//处理进位
	for (int j = c.size() - 1; j > 0; j--)
	{
		if (c[j] >= base)
		{
			c[j - 1] += c[j] / base;
			c[j] %= base;
		}
	}
	for (int j = c.size() - 1; j > 0; j--)
	{
		if (c[j] < 0) {
			c[j] = 10 + c[j];
			c[j - 1] = c[j - 1] - 1;
		}
	}

	//去掉多余的零
	c.pop_back();
	int i = 0;
	if (c[0] == 0) { i++; }
	if (changeflag == true) {
		c[0] = -c[0];
	}
	vector<float> result;
	result.insert(result.end(), c.begin(), c.end());
	//输出最终结果，改了进制需要改这里的输出方式，比如百进制是"%02d",千进制是"%03d"
	return result;
}
vector<float> addmain(vector<float> a, vector<float> b)
{
	int N = a.size();
	vector<float> c(N + 1);
	float *dev_a, *dev_b, *dev_c;
	hipMalloc((void **)&dev_a, N * sizeof(float));
	hipMalloc((void **)&dev_b, N * sizeof(float));
	hipMalloc((void **)&dev_c, (N + 1) * sizeof(float));
	hipMemcpy(dev_a, a.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c.data(), (N + 1) * sizeof(float), hipMemcpyHostToDevice);
	add << <N, 1 >> >(dev_a, dev_b, dev_c);//<N,1>,第一个参数N代表block的数量，第二个参数1代表每个block中thread的数量
	hipMemcpy(&c[0], dev_c, (N + 1) * sizeof(int), hipMemcpyDeviceToHost);
	return c;
}
__global__ void add(const float* a, const float*b, float* c)
{
	int tid = blockIdx.x;//blockIdx是一个内置变量，blockIdx.x代表这是一个2维索引
	c[tid] = a[tid] + b[tid];
}
vector<float> submain(vector<float> a, vector<float> b)
{
	int N = a.size();
	vector<float> c(N + 1);
	float *dev_a, *dev_b, *dev_c;
	hipMalloc((void **)&dev_a, N * sizeof(float));
	hipMalloc((void **)&dev_b, N * sizeof(float));
	hipMalloc((void **)&dev_c, (N + 1) * sizeof(float));
	hipMemcpy(dev_a, a.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c.data(), (N + 1) * sizeof(float), hipMemcpyHostToDevice);
	subVector << <N, 1 >> >(dev_a, dev_b, dev_c);//<N,1>,第一个参数N代表block的数量，第二个参数1代表每个block中thread的数量
	hipMemcpy(&c[0], dev_c, (N + 1) * sizeof(int), hipMemcpyDeviceToHost);
	return c;
}
__global__ void subVector(const float* a, const float*b, float* c)
{
	int tid = blockIdx.x;//blockIdx是一个内置变量，blockIdx.x代表这是一个2维索引
	c[tid] = a[tid] - b[tid];
}

node* sum(node *p, node *u, int n)
{
	node *a = new node;
	node *b = new node;
	node *l = new node;
	node *h = new node;
	h->next = NULL;
	l = h;
	a = p->next;
	b = u->next;
	while (a != NULL&&b != NULL)
	{
		if (a->zhi>b->zhi)
		{
			node *m = new node;
			m->zhi = a->zhi;
			m->xi = a->xi;
			m->next = NULL;
			l->next = m;
			l = m;
			a = a->next;
		}
		else if (a->zhi<b->zhi)
		{
			node *m = new node;
			m->zhi = b->zhi;
			m->xi = b->xi;
			m->next = NULL;
			l->next = m;
			l = m;
			b = b->next;
		}
		else
		{
			if ((a->xi.size() + b->xi.size()) != 0)
			{
				node *m = new node;
				m->xi = addInteger(a->xi, b->xi);
				m->zhi = a->zhi;
				m->next = NULL;
				l->next = m;
				l = m;
				b = b->next;
				a = a->next;
			}
			else
			{
				b = b->next;
				a = a->next;
			}
		}
	}
	if (a == NULL&&b != NULL)
	{
		while (b != NULL)
		{
			node *m = new node;
			m->xi = b->xi;
			m->zhi = b->zhi;
			m->next = NULL;
			l->next = m;
			l = m;
			b = b->next;
		}
	}
	else if (a != NULL&&b == NULL)
	{
		while (a != NULL)
		{
			node *m = new node;
			m->xi = a->xi;
			m->zhi = a->zhi;
			m->next = NULL;
			l->next = m;
			l = m;
			a = a->next;
		}
	}
	return h;
}
node *sub(node *p, node *u, int n)
{
	node *a = new node;
	node *b = new node;
	node *l = new node;
	node *h = new node;
	h->next = NULL;
	l = h;
	a = p->next;
	b = u->next;
	while (a != NULL&&b != NULL)
	{
		if (a->zhi>b->zhi)
		{
			node *m = new node;
			m->zhi = a->zhi;
			m->xi = a->xi;
			m->next = NULL;
			l->next = m;
			l = m;
			a = a->next;
		}
		else if (a->zhi<b->zhi)
		{
			node *m = new node;
			//b->xi[0]=-b->xi[0];
			m->zhi = b->zhi;
			m->xi = b->xi;
			m->xi[0] = -m->xi[0];
			m->next = NULL;
			l->next = m;
			l = m;
			b = b->next;
		}
		else
		{
			if ((a->xi.size() + b->xi.size()) != 0)
			{
				node *m = new node;
				//b->xi[0]=-b->xi[0];
				m->xi = b->xi;
				m->xi[0] = -m->xi[0];
				m->xi = addInteger(a->xi, m->xi);
				m->zhi = a->zhi;
				m->next = NULL;
				l->next = m;
				l = m;
				b = b->next;
				a = a->next;
			}
			else
			{
				b = b->next;
				a = a->next;
			}
		}
	}
	if (a == NULL&&b != NULL)
	{
		while (b != NULL)
		{
			node *m = new node;
			m->xi = b->xi;
			m->xi[0] = -m->xi[0];
			m->zhi = b->zhi;
			m->next = NULL;
			l->next = m;
			l = m;
			b = b->next;
		}
	}
	else if (a != NULL&&b == NULL)
	{
		while (a != NULL)
		{
			node *m = new node;
			m->xi = a->xi;
			m->zhi = a->zhi;
			m->next = NULL;
			l->next = m;
			l = m;
			a = a->next;
		}
	}
	return h;
}

