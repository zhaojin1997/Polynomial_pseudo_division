#include "hip/hip_runtime.h"
#include "mult.h"
node* mult(node *p, node *u, int n)
{
	node *a = p->next, *b = u->next, *d;
	node *t = new node;
	t->next = NULL;
	d = t;
	node *y = new node;

	/*node *r = new node;
	r->zhi.clear();
	r->xi.clear();
	for (int i = 0; i < n; i++) {
	r->zhi.insert(r->zhi.end(), 0);
	}
	r->next = NULL;
	y->next = r;*/
	y->next = NULL;
	while (a != NULL)
	{
		b = u->next;
		d = t;
		while (b != NULL)
		{
			node *c = new node;
			c->zhi.clear();
			for (int i = 0; i<n; i++) {
				c->zhi.insert(c->zhi.end(), (a->zhi[i] + b->zhi[i]));
			}
			c->xi = multi(a->xi, b->xi);
			c->next = NULL;
			d->next = c;
			d = c;
			b = b->next;
		}
		y = sum(y, t, n);
		a = a->next;
	}
	return y;
}
__global__ void ConvertToInt(hipfftReal *a, int size)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	int* b = (int*)a;
	for (int i = threadID; i < size; i += numThreads)
		b[i] = static_cast<int>(round(a[i]));
}
vector<float> multi(vector<float> aa, vector<float> bb) {
	int lena = aa.size();
	int lenb = bb.size();
	float aflag = aa[0];
	float bflag = bb[0];
	//	cout << "aflag:" << aflag << ",bflag:" << bflag << endl;
	if (aflag < 0)aa[0] = -aa[0];
	if (bflag < 0)bb[0] = -bb[0];
	vector<float> a(lenb);
	vector<float> b(lena);
	//在vector首部执行大量insert操作会严重影响性能，因此转换为在尾部执行
	a.insert(a.end(), aa.begin(), aa.end());
	b.insert(b.end(), bb.begin(), bb.end());
	//printf(".....................do multiply............................\n");
	//设置进制
	vector<int> c = multiply(a, b);
	const int base = 10;
	//处理进位
	for (int j = c.size() - 1; j > 0; j--)
	{
		if (c[j] >= base)
		{
			c[j - 1] += c[j] / base;
			c[j] %= base;
		}
	}

	//去掉多余的零
	c.pop_back();
	int i = 0;
	if (c[0] == 0) { i++; }
	if ((aflag*bflag) < 0)c[0] = -c[0];
	vector<float> result;
	result.insert(result.end(), c.begin(), c.end());
	//输出最终结果，改了进制需要改这里的输出方式，比如百进制是"%02d",千进制是"%03d"
	return result;
}
vector<int> multiply(const vector<float> &a, const vector<float> &b)
{
	//时域转换频域后进行点乘
	const int NX = a.size();
	hipfftHandle plan_a, plan_b, plan_c;
	hipfftComplex *data_a, *data_b;
	vector<int> c(a.size());

	//分配显卡内存并初始化，这里假设sizeof(int)==sizeof(float), sizeof(hipfftComplex)==2*sizeof(float)
	hipMalloc((void**)&data_a, sizeof(hipfftComplex) * (NX / 2 + 1) * BATCH);
	hipMalloc((void**)&data_b, sizeof(hipfftComplex) * (NX / 2 + 1) * BATCH);
	hipMemcpy(data_a, a.data(), sizeof(float) * a.size(), hipMemcpyHostToDevice);
	hipMemcpy(data_b, b.data(), sizeof(float) * b.size(), hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) { fprintf(stderr, "Cuda error: Failed to allocate\n"); return c; }

	if (hipfftPlan1d(&plan_a, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }
	if (hipfftPlan1d(&plan_b, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }
	if (hipfftPlan1d(&plan_c, NX, HIPFFT_C2R, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }

	//把A(x)转换到频域
	if (hipfftExecR2C(plan_a, (hipfftReal*)data_a, data_a) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
		return c;
	}

	//把B(x)转换到频域
	if (hipfftExecR2C(plan_b, (hipfftReal*)data_b, data_b) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
		return c;
	}

	//点乘
	ComplexPointwiseMulAndScale << <NX / 256 + 1, 256 >> >(data_a, data_b, NX);

	//把C(x)转换回时域
	if (hipfftExecC2R(plan_c, data_b, (hipfftReal*)data_b) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecC2R Forward failed");
		return c;
	}

	//将浮点数的结果转换为整数
	ConvertToInt << <NX / 256 + 1, 256 >> >((hipfftReal*)data_b, NX);

	if (hipDeviceSynchronize() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
		return c;
	}

	hipMemcpy(&c[0], data_b, sizeof(float) * b.size(), hipMemcpyDeviceToHost);

	hipfftDestroy(plan_a);
	hipfftDestroy(plan_b);
	hipfftDestroy(plan_c);
	hipFree(data_a);
	hipFree(data_b);
	return c;
}
__global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, hipfftComplex *b, int size)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	float scale = 1.0f / (float)size;
	hipfftComplex c;
	for (int i = threadID; i < size; i += numThreads)
	{
		c = hipCmulf(a[i], b[i]);//a*b
		b[i] = make_hipFloatComplex(scale*hipCrealf(c), scale*hipCimagf(c));//分别对复数的实部和虚部乘以系数
	}
}

