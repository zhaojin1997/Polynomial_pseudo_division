#include "hip/hip_runtime.h"
#include "support.h"
#include "struct.h"
#include "add.h"
#include "mult.h"
#include "device.h"
#include "ComputeCpu.h"


int main(int argc, char *argv[])
{
printf("请输入变量个数:");
	int temNum = 0;
	cin >> temNum;
	printf("请输入变量名：");
	char* temName = (char*)malloc(temNum * sizeof(char));
	for (int i = 0; i<temNum; i++)
	{
		cin >> temName[i];
	}
	cout << "\n---------------------------------------------------" << endl;
	cout << "请按提示输入多项式A:" << endl;;
	struct node *a = new node;//多项式A
	input(a, temNum, temName);
	cout << "\n---------------------------------------------------" << endl;
	cout << "请按提示输入多项式B:" << endl;;
	struct node *b = new node;//多项式B
	input(b, temNum, temName);
	cout << "\n-----------------------DEVICE--------------------------" << endl;
	hipEvent_t start2;
	hipEventCreate(&start2);
	hipEvent_t stop2;
	hipEventCreate(&stop2);
	hipEventRecord(start2, NULL);

	device(a, b, temNum, temName);

	hipEventRecord(stop2, NULL);
	hipEventSynchronize(stop2);
	float msecTotal2 = 0.0f;
	hipEventElapsedTime(&msecTotal2, start2, stop2);
	printf("DEVICE CUDA Time:%f ms\n", msecTotal2);

	clock_t begin2, end2;
	begin2 = clock();

	devicecpu(a, b, temNum, temName);

	end2 = clock();
	float time2 = ((float)(end2 - begin2));
	printf("DEVICE CPU: %f ms\n", time2);
	printf("DEVICE Spped up:%f\n", time2 / msecTotal2);
	return 0;
}
